#include "hip/hip_runtime.h"
/*
 * phase_oscillator_ensemble.cu
 *
 * The example how the phase_oscillator ensemble can be implemented using CUDA and thrust
 *
 *  Created on: July 15, 2011
 *      Author: karsten
 */


#include <iostream>
#include <cmath>
#include <utility>

#include <thrust/device_vector.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>

#include <boost/numeric/odeint.hpp>
#include <boost/numeric/odeint/external/thrust/thrust_algebra.hpp>
#include <boost/numeric/odeint/external/thrust/thrust_operations.hpp>
#include <boost/numeric/odeint/external/thrust/thrust_resize.hpp>

#include <boost/random.hpp>



using namespace std;

using namespace boost::numeric::odeint;

//change this to float if your device does not support double computation
typedef double value_type;

//change this to host_vector< ... > of you want to run on CPU
typedef thrust::device_vector< value_type > state_type;
typedef thrust::device_vector< size_t > index_vector_type;
// typedef thrust::host_vector< value_type > state_type;
// typedef thrust::host_vector< size_t > index_vector_type;


struct calc_mean_field
{
    struct sin_functor : public thrust::unary_function< value_type , value_type >
    {
        __host__ __device__
        value_type operator()( value_type x) const
        {
            return sin( x );
        }
    };

    struct cos_functor : public thrust::unary_function< value_type , value_type >
    {
        __host__ __device__
        value_type operator()( value_type x) const
        {
            return cos( x );
        }
    };

    std::pair< value_type , value_type > get_mean( const state_type &x ) const
    {
        value_type sin_sum = thrust::reduce(
                thrust::make_transform_iterator( x.begin() , sin_functor() ) ,
                thrust::make_transform_iterator( x.end() , sin_functor() ) );
        value_type cos_sum = thrust::reduce(
                thrust::make_transform_iterator( x.begin() , cos_functor() ) ,
                thrust::make_transform_iterator( x.end() , cos_functor() ) );

        cos_sum /= value_type( x.size() );
        sin_sum /= value_type( x.size() );

        value_type K = sqrt( cos_sum * cos_sum + sin_sum * sin_sum );
        value_type Theta = atan2( sin_sum , cos_sum );

        return std::make_pair( K , Theta );
    }
};

class phase_oscillator_ensemble
{

public:

    struct sys_functor
    {
        value_type m_K , m_Theta , m_epsilon;
        sys_functor( value_type K , value_type Theta , value_type epsilon )
        : m_K( K ) , m_Theta( Theta ) , m_epsilon( epsilon ) { }

        template< class Tuple >
        __host__ __device__
        void operator()( Tuple t )
        {
            thrust::get<2>(t) = thrust::get<1>(t) + m_epsilon * m_K * sin( m_Theta - thrust::get<0>(t) );
        }
    };

    phase_oscillator_ensemble( size_t N , value_type g = 1.0 , value_type epsilon = 1.0 )
        : m_omega() , m_N( N ) , m_epsilon( epsilon )
    {
        create_frequencies( g );
    }

    void create_frequencies( value_type g )
    {
        boost::mt19937 rng;
        boost::cauchy_distribution< value_type > cauchy( 0.0 , g );
        boost::variate_generator< boost::mt19937&, boost::cauchy_distribution< value_type > > gen( rng , cauchy );
        vector< value_type > omega( m_N );
        generate( omega.begin() , omega.end() , gen );
        m_omega = omega;
    }

    void set_epsilon( value_type epsilon ) { m_epsilon = epsilon; }

    value_type get_epsilon( void ) const { return m_epsilon; }

    void operator() ( const state_type &x , state_type &dxdt , const value_type dt ) const
    {
        calc_mean_field mean_field_calculator;
        std::pair< value_type , value_type > mean_field = mean_field_calculator.get_mean( x );

        thrust::for_each(
                thrust::make_zip_iterator( thrust::make_tuple( x.begin() , m_omega.begin() , dxdt.begin() ) ),
                thrust::make_zip_iterator( thrust::make_tuple( x.end() , m_omega.end() , dxdt.end()) ) ,
                sys_functor( mean_field.first , mean_field.second , m_epsilon )
                );
    }

private:

    state_type m_omega;
    const size_t m_N;
    value_type m_epsilon;
};


//[ phase_oscillator_ensemble_observer
struct statistics_observer
{
    value_type m_K_mean;
    size_t m_count;

    statistics_observer( void )
    : m_K_mean( 0.0 ) , m_count( 0 ) { }

    template< class State >
    void operator()( const State &x , value_type t )
    {
        calc_mean_field mean_field_calculator;
        std::pair< value_type , value_type > mean = mean_field_calculator.get_mean( x );
        m_K_mean += mean.first;
        ++m_count;
    }

    value_type get_K_mean( void ) const { return ( m_count != 0 ) ? m_K_mean / value_type( m_count ) : 0.0 ; }

    void reset( void ) { m_K_mean = 0.0; m_count = 0; }
};
//]



// const size_t N = 16384 * 128;
const size_t N = 16384;
const value_type pi = 3.1415926535897932384626433832795029;
const value_type dt = 0.1;

int main( int arc , char* argv[] )
{
    boost::mt19937 rng;
    boost::uniform_real< value_type > unif( 0.0 , 2.0 * pi );
    boost::variate_generator< boost::mt19937&, boost::uniform_real< value_type > > gen( rng , unif );

    // vectors for host and device
    vector< value_type > x_host( N );
    state_type x( N );


    //create error stepper
    runge_kutta4< state_type , value_type , state_type , value_type , thrust_algebra , thrust_operations > stepper;
    phase_oscillator_ensemble ensemble( N , 1.0 );
    statistics_observer obs;

    for( value_type epsilon = 0.0 ; epsilon < 5.0 ; epsilon += 0.1 )
    {
        ensemble.set_epsilon( epsilon );
        obs.reset();

        // start with random initial conditions
        generate( x_host.begin() , x_host.end() , gen );
        x = x_host;

        // calculate some transients steps
        integrate_const( stepper , boost::ref( ensemble ) , x , 0.0 , 10.0 , dt );

        // integrate and compute the statistics
        integrate_const( stepper , boost::ref( ensemble ) , x , 0.0 , 100.0 , dt , boost::ref( obs ) );
        cout << epsilon << "\t" << obs.get_K_mean() << endl;
    }
}
