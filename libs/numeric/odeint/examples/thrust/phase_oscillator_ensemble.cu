#include "hip/hip_runtime.h"
/*
 * phase_oscillator_example.cu
 *
 * The example how the phase_oscillator ensemble can be implemented using CUDA and thrust
 *
 *  Created on: July 15, 2011
 *      Author: karsten
 */


#include <iostream>
#include <cmath>
#include <cstdlib>
#include <ctime>

#include <thrust/device_vector.h>

#include <boost/numeric/odeint.hpp>
#include <boost/numeric/odeint/external/thrust/thrust_algebra.hpp>
#include <boost/numeric/odeint/external/thrust/thrust_operations.hpp>
#include <boost/numeric/odeint/external/thrust/thrust_resize.hpp>

using namespace std;

using namespace boost::numeric::odeint;

//change this to float if your device does not support double computation
typedef double value_type;

//change this to host_vector< ... > of you want to run on CPU
typedef thrust::device_vector< value_type > state_type;
typedef thrust::device_vector< size_t > index_vector_type;
//typedef thrust::host_vector< value_type > state_type;
//typedef thrust::host_vector< size_t > index_vector_type;




/*
 * This implements the rhs of the dynamical equation:
 * \phi'_0 = \omega_0 + sin( \phi_1 - \phi_0 )
 * \phi'_i  = \omega_i + sin( \phi_i+1 - \phi_i ) + sin( \phi_i - \phi_i-1 )
 * \phi'_N-1 = \omega_N-1 + sin( \phi_N-1 - \phi_N-2 )
 */
class phase_oscillator_ensemble
{

public:

    struct mean_field_functor
    {
        template< class Tuple >
        __host__ __device__
        void operator()( Tuple t )
        {

        }
    };

    struct sys_functor
    {
        value_type m_K , m_Theta;
        sys_functor( value_type K , value_type Theta )
        : m_K( K ) , m_Theta( Theta ) { }

        template< class Tuple >
        __host__ __device__
        void operator()( Tuple t )
        {
//            thrust::get<4>(t) = omega + sin( phi_right - phi ) + sin( phi - phi_left );
        }
    };

    phase_oscillators( state_type &omega )
        : m_omega( omega ) , m_N( omega.size() ) , m_prev( m_N ) , m_next( m_N )
    {
        // build indices pointing to left and right neighbours
        thrust::counting_iterator<size_t> c( 0 );
        thrust::copy( c , c+m_N-1 , m_prev.begin()+1 );
        m_prev[0] = 0; // m_prev = { 0 , 0 , 1 , 2 , 3 , ... , N-1 }

        thrust::copy( c+1 , c+m_N , m_next.begin() );
        m_next[m_N-1] = m_N-1; // m_next = { 1 , 2 , 3 , ... , N-1 , N-1 }

        /*thrust::copy( m_prev.begin() , m_prev.end() ,
                    std::ostream_iterator< size_t >(std::cout, " ") );
        std::cout << std::endl;*/
    }



    void operator() ( const state_type &x , state_type &dxdt , const value_type dt )
    {
        thrust::for_each(
                thrust::make_zip_iterator(
                        thrust::make_tuple(
                                x.begin() ,
                                thrust::make_permutation_iterator( x.begin() , m_prev.begin() ) ,
                                thrust::make_permutation_iterator( x.begin() , m_next.begin() ) ,
                                m_omega.begin() ,
                                dxdt.begin()
                                ) ),
                thrust::make_zip_iterator(
                        thrust::make_tuple(
                                x.end() ,
                                thrust::make_permutation_iterator( x.begin() , m_prev.end() ) ,
                                thrust::make_permutation_iterator( x.begin() , m_next.end() ) ,
                                m_omega.end() ,
                                dxdt.end()) ) ,
                sys_functor()
                );
    }

private:
    const state_type &m_omega;
    const size_t m_N;
    index_vector_type m_prev;
    index_vector_type m_next;
};


const size_t N = 16;
const value_type epsilon = 6.0/(N*N); // should be < 8/N^2 to see phase locking

int main( int arc , char* argv[] )
{
    srand( time(NULL) );
    // create initial conditions on host:
    vector< value_type > x_host( N );
    //create omegas on host
    vector< value_type > omega_host( N );
    for( size_t i=0 ; i<N ; ++i )
    {
        x_host[i] = 2.0*3.14159265*(double)(rand())/RAND_MAX;
        omega_host[i] = (N-i)*epsilon; // decreasing frequencies
    }

    //copy to device
    state_type x = x_host;
    state_type omega = omega_host;

    //create error stepper
    explicit_rk4< state_type , value_type , state_type , value_type ,
                  thrust_algebra , thrust_operations , adjust_size_initially_tag  > stepper;

    phase_oscillators sys( omega );

    value_type t = 0.0;
    const value_type dt = 0.1;
    while( t < 10.0 )
    {
        stepper.do_step( sys , x , t , dt );
        t += dt;
    }

    /**ToDo: use integrate functions, maybe with algebra_dispatcher */

    //perform integration using standard Runge-Kutta-Cash-Carp Stepper and error bounds ~ 1E-6
    //integrate_const( phase_oscillators(omega) , x , 0.0 , 100.0 , 0.1 );

    thrust::copy( x.begin() , x.end() ,
            std::ostream_iterator< value_type >(std::cout, " ") );
    std::cout << std::endl;
}
